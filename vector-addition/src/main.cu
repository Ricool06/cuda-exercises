
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int numElements = 1<<20; // 1M elements

  float *x, *y;
  hipMallocManaged(&x, numElements * sizeof(float));
  hipMallocManaged(&y, numElements * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < numElements; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(numElements, x, y);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < numElements; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
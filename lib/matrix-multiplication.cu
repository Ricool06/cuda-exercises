#include "hip/hip_runtime.h"
#include "matrix-multiplication.cuh"
#include "handle-error.cuh"

__global__
void multiplyMatricesKernel(Matrix resultMatrix, Matrix matrixOne, Matrix matrixTwo) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t column = blockIdx.x * blockDim.x + threadIdx.x;
    size_t resultIndex = (row * resultMatrix.width) + column;

    if (row < matrixOne.height && column < matrixTwo.width) {
        
        int result = 0;
        for(size_t i = 0; i < matrixOne.width; i++) {
            result += (matrixOne.elements[(row * matrixOne.width) + i]) * (matrixTwo.elements[(i * matrixTwo.width) + column]);
        }

        resultMatrix.elements[resultIndex] = result;
    }
}

void multiplyMatrices(Matrix resultMatrix, Matrix matrixOne, Matrix matrixTwo) {
    static const size_t tileSize = 32;

    size_t resultMatrixSizeInBytes = resultMatrix.height * resultMatrix.width * sizeof(int);
    size_t matrixOneSizeInBytes = matrixOne.height * matrixOne.width * sizeof(int);
    size_t matrixTwoSizeInBytes = matrixTwo.height * matrixTwo.width * sizeof(int);

    Matrix dResultMatrix(matrixOne.height, matrixTwo.width, nullptr);
    Matrix dMatrixOne(matrixOne.height, matrixOne.width, nullptr);
    Matrix dMatrixTwo(matrixTwo.height, matrixTwo.width, nullptr);

    checkCudaCall(hipMalloc(&dResultMatrix.elements, resultMatrixSizeInBytes));
    checkCudaCall(hipMalloc(&dMatrixOne.elements, matrixOneSizeInBytes));
    checkCudaCall(hipMalloc(&dMatrixTwo.elements, matrixTwoSizeInBytes));

    checkCudaCall(hipMemcpy(dMatrixOne.elements, matrixOne.elements, matrixOneSizeInBytes, hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(dMatrixTwo.elements, matrixTwo.elements, matrixTwoSizeInBytes, hipMemcpyHostToDevice));

    size_t xBlocksCount = (resultMatrix.width + tileSize - 1) / tileSize;
    size_t yBlocksCount = (resultMatrix.height + tileSize - 1) / tileSize;
    dim3 gridDimensions(xBlocksCount, yBlocksCount);
    dim3 blockDimensions(tileSize, tileSize);

    multiplyMatricesKernel<<<gridDimensions, blockDimensions>>>(dResultMatrix, dMatrixOne, dMatrixTwo);
    checkCudaCall(hipDeviceSynchronize());

    checkCudaCall(hipMemcpy(resultMatrix.elements, dResultMatrix.elements, resultMatrixSizeInBytes, hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(dMatrixOne.elements));
    checkCudaCall(hipFree(dMatrixTwo.elements));
    checkCudaCall(hipFree(dResultMatrix.elements));
}
#include "hip/hip_runtime.h"
#include "handle-error.cuh"

__global__
void addVectorsKernel(int *resultVector, std::size_t length, int *vectorOne, int *vectorTwo) {
    for (std::size_t i = threadIdx.x; i < length; i += blockDim.x)
        resultVector[i] = vectorOne[i] + vectorTwo[i];
};

void addVectors(int *resultVector, std::size_t length, int *vectorOne, int *vectorTwo) {
    int *deviceVectorOne, *deviceVectorTwo, *deviceResultVector;
    size_t arraySizeInBytes = length * sizeof(int);

    checkCudaCall(hipMalloc(&deviceVectorOne, arraySizeInBytes));
    checkCudaCall(hipMalloc(&deviceVectorTwo, arraySizeInBytes));
    checkCudaCall(hipMalloc(&deviceResultVector, arraySizeInBytes));

    checkCudaCall(hipMemcpy(deviceVectorOne, vectorOne, arraySizeInBytes, hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceVectorTwo, vectorTwo, arraySizeInBytes, hipMemcpyHostToDevice));

    std::size_t blockSize = 256;
    std::size_t numBlocks = (length + blockSize - 1) / blockSize;

    addVectorsKernel<<<numBlocks, blockSize>>>(deviceResultVector, length, deviceVectorOne, deviceVectorTwo);

    checkCudaCall(hipFree(deviceVectorOne));
    checkCudaCall(hipFree(deviceVectorTwo));

    checkCudaCall(hipMemcpy(resultVector, deviceResultVector, arraySizeInBytes, hipMemcpyDeviceToHost));
};


#include <hip/hip_runtime.h>
__global__
void addVectorsKernel(int *resultVector, std::size_t length, int *vectorOne, int *vectorTwo) {
    for (std::size_t i = threadIdx.x; i < length; i += blockDim.x)
        resultVector[i] = vectorOne[i] + vectorTwo[i];
};

void addVectors(int *resultVector, std::size_t length, int *vectorOne, int *vectorTwo) {
    int *deviceVectorOne, *deviceVectorTwo, *deviceResultVector;
    size_t arraySizeInBytes = length * sizeof(int);

    hipMalloc(&deviceVectorOne, arraySizeInBytes);
    hipMalloc(&deviceVectorTwo, arraySizeInBytes);
    hipMalloc(&deviceResultVector, arraySizeInBytes);

    hipMemcpy(deviceVectorOne, vectorOne, arraySizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceVectorTwo, vectorTwo, arraySizeInBytes, hipMemcpyHostToDevice);

    std::size_t blockSize = 256;
    std::size_t numBlocks = (length + blockSize - 1) / blockSize;

    addVectorsKernel<<<numBlocks, blockSize>>>(deviceResultVector, length, deviceVectorOne, deviceVectorTwo);

    hipFree(deviceVectorOne);
    hipFree(deviceVectorTwo);

    hipMemcpy(resultVector, deviceResultVector, arraySizeInBytes, hipMemcpyDeviceToHost);
};

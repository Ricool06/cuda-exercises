
#include <hip/hip_runtime.h>
#include <stdio.h>

static void checkCudaCall_f(hipError_t hipError_t, const char* file, int line) {
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "ERROR: CUDA call failed in file: %s at line %d\n", file, line);
        exit(hipError_t);
    }
};

void checkCudaCall(hipError_t hipError_t) {
    checkCudaCall_f(hipError_t, __FILE__, __LINE__);
};

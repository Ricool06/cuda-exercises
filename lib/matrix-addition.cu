#include "hip/hip_runtime.h"
#include "matrix-addition.cuh"
#include "handle-error.cuh"

__global__
void addMatricesKernel(Matrix resultMatrix, Matrix matrixOne, Matrix matrixTwo) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < resultMatrix.height * resultMatrix.width) {
        resultMatrix.elements[id] = matrixOne.elements[id] + matrixTwo.elements[id];
    }
}

void addMatrices(Matrix resultMatrix, Matrix matrixOne, Matrix matrixTwo) {
    size_t resultMatrixSize = resultMatrix.height * resultMatrix.width;
    size_t resultMatrixSizeInBytes = resultMatrixSize * sizeof(int);

    Matrix dResultMatrix = Matrix(resultMatrix.height, resultMatrix.width, nullptr);
    Matrix dMatrixOne = Matrix(matrixOne.height, matrixOne.width, nullptr);
    Matrix dMatrixTwo = Matrix(matrixTwo.height, matrixTwo.width, nullptr);

    checkCudaCall(hipMalloc(&dResultMatrix.elements, resultMatrixSizeInBytes));
    checkCudaCall(hipMalloc(&dMatrixOne.elements, resultMatrixSizeInBytes));
    checkCudaCall(hipMalloc(&dMatrixTwo.elements, resultMatrixSizeInBytes));

    checkCudaCall(hipMemcpy(dMatrixOne.elements, matrixOne.elements, resultMatrixSizeInBytes, hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(dMatrixTwo.elements, matrixTwo.elements, resultMatrixSizeInBytes, hipMemcpyHostToDevice));

    size_t blockSize = 1024;
    size_t numBlocks = (resultMatrixSize + blockSize - 1) / blockSize;

    addMatricesKernel<<<numBlocks, blockSize>>>(dResultMatrix, dMatrixOne, dMatrixTwo);

    checkCudaCall(hipFree(dMatrixOne.elements));
    checkCudaCall(hipFree(dMatrixTwo.elements));

    checkCudaCall(hipMemcpy(resultMatrix.elements, dResultMatrix.elements, resultMatrixSizeInBytes, hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(dResultMatrix.elements));
}
